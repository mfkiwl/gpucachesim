#include "hip/hip_runtime.h"
#include <cstdarg>
#include <stdint.h>
#include <stdio.h>

#include "utils/channel.hpp"
#include "utils/utils.h"

// contains definition of the mem_access_t structure
#include "common.h"

// Instrumentation function that we want to inject.
// Please note the use of extern "C" __device__ __noinline__
// to prevent "dead"-code elimination by the compiler.
extern "C" __device__ __noinline__ void
instrument_inst(int pred, int instr_opcode_id, uint32_t instr_offset,
                uint32_t instr_idx, int instr_predicate_num,
                bool instr_predicate_is_neg, bool instr_predicate_is_uniform,
                uint32_t instr_mem_space, bool instr_is_load,
                bool instr_is_store, bool instr_is_extended, uint64_t addr,
                uint64_t grid_launch_id, uint64_t pchannel_dev) {

  /* if thread is predicated off, return */
  if (!pred) {
    return;
  }

  int active_mask = __ballot_sync(__activemask(), 1);
  const int laneid = get_laneid();
  const int first_laneid = __ffs(active_mask) - 1;

  mem_access_t ma;

  /* collect memory address information from other threads */
  for (int i = 0; i < 32; i++) {
    ma.addrs[i] = __shfl_sync(active_mask, addr, i);
  }

  int4 cta = get_ctaid();
  ma.grid_launch_id = grid_launch_id;
  ma.cta_id_x = cta.x;
  ma.cta_id_y = cta.y;
  ma.cta_id_z = cta.z;
  ma.warp_id = get_warpid();
  ma.instr_opcode_id = instr_opcode_id;
  ma.instr_offset = instr_offset;
  ma.instr_idx = instr_idx;
  ma.instr_predicate_num = instr_predicate_num;
  ma.instr_predicate_is_neg = instr_predicate_is_neg;
  ma.instr_predicate_is_uniform = instr_predicate_is_uniform;
  ma.instr_mem_space = instr_mem_space;
  ma.instr_is_load = instr_is_load;
  ma.instr_is_store = instr_is_store;
  ma.instr_is_extended = instr_is_extended;

  /* first active lane pushes information on the channel */
  if (first_laneid == laneid) {
    ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;
    channel_dev->push(&ma, sizeof(mem_access_t));
  }
}
