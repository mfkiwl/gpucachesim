#include <assert.h>
#include <cstdlib>
#include <stdint.h>
#include <stdio.h>

#include "hip/hip_runtime.h"

#define CUDA_SAFECALL(call)                                                    \
  {                                                                            \
    call;                                                                      \
    hipError_t err = hipGetLastError();                                        \
    if (hipSuccess != err) {                                                  \
      fprintf(stderr,                                                          \
              "Cuda error in function '%s' file '%s' in line %i : %s.\n",      \
              #call, __FILE__, __LINE__, hipGetErrorString(err));             \
      fflush(stderr);                                                          \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

const bool USE_COMPRESSION = false;

const int KB = 1024;

// have 96 KB shared memory, sizeof(unsigned)
// const static size_t SHMEM_SIZE_BYTES = 0xC000;

// have only 48KB shared mem because we test the 16KB L1?
// const size_t SHMEM_SIZE_BYTES = 16 * (1 << 10);
// const int ITER_SIZE = 512;
// const int ITER_SIZE = 4 * 1024;
// const int ITER_SIZE = 8 * 1024;
// const int ITER_SIZE = 6 * 1024;
// const int ITER_SIZE = (SHMEM_SIZE_BYTES / 2) / sizeof(unsigned int);

// const int ITER_SIZE = 6 * 1024;
const int ITER_SIZE = ((48 * KB) / 2) / sizeof(uint32_t);

__global__ void global_latency_l1_data(unsigned int *array, int array_length,
                                       unsigned int *duration,
                                       unsigned int *index,
                                       size_t warmup_iterations) {
  unsigned int start_time, end_time;
  uint32_t warmup_j = 0;
  uint32_t j = 0;

  __shared__ uint32_t s_tvalue[ITER_SIZE];
  __shared__ uint32_t s_index[ITER_SIZE];

  for (size_t k = 0; k < ITER_SIZE; k++) {
    s_index[k] = 0;
    s_tvalue[k] = 0;
  }

  for (int k = (int)warmup_iterations * -ITER_SIZE; k < ITER_SIZE; k++) {
    if (k >= 0) {
      start_time = clock();
      j = array[j];
      s_index[k] = j;
      end_time = clock();

      s_tvalue[k] = end_time - start_time;
    } else {
      j = array[j];
      // warmup_j = array[warmup_j];
    }
  }

  array[array_length] = j;
  array[array_length + 1] = array[j];

  for (size_t k = 0; k < ITER_SIZE; k++) {
    index[k] = s_index[k];
    duration[k] = s_tvalue[k];
  }
}

// can store latencies 0-100, 100-200, .. 1500-1600
// const static unsigned int LATENCY_BIN_COUNT = 16;
const static unsigned int LATENCY_BIN_SIZE = 16;
// const static unsigned int LATENCY_BIN_BITS = 4; //
// const static unsigned int LATENCIES_PER_TVALUE = 8;
//
// constexpr unsigned floorlog2(unsigned x) {
//   return x == 1 ? 0 : 1 + floorlog2(x >> 1);
// }
//
// constexpr unsigned ceillog2(unsigned x) {
//   return x == 1 ? 0 : floorlog2(x - 1) + 1;
// }
//
// static_assert(ceillog2(LATENCY_BIN_COUNT) == LATENCY_BIN_BITS,
//               "correct latency bin bits");
// static_assert(sizeof(uint32_t) * 8 / LATENCY_BIN_BITS ==
// LATENCIES_PER_TVALUE,
//               "correct latencies per tvalue");
// static_assert(sizeof(uint32_t) * 8 == 32, "uint32 is 32 bits");
// static_assert(CHAR_BIT == 8, "have 8 bits per byte");

// const int ITER_SIZE_COMPRESSED = 12 * 1024;
const int ITER_SIZE_COMPRESSED = (48 * KB) / sizeof(uint8_t);

__global__ void global_latency_compressed(unsigned int *array, int array_length,
                                          unsigned int *duration,
                                          unsigned int *index,
                                          size_t warmup_iterations) {
  unsigned int start_time, end_time, dur;
  uint32_t j = 0;

  __shared__ uint8_t s_tvalue[ITER_SIZE_COMPRESSED];
  // __shared__ uint32_t s_index[ITER_SIZE];

  for (size_t k = 0; k < ITER_SIZE_COMPRESSED; k++) {
    // s_index[k] = 0;
    s_tvalue[k] = 0;
  }

  for (int k = (int)warmup_iterations * -ITER_SIZE_COMPRESSED;
       k < ITER_SIZE_COMPRESSED; k++) {
    if (k >= 0) {
      start_time = clock();
      j = array[j];
      // s_index[k] = j;
      s_tvalue[k] = j;
      end_time = clock();

      dur = (end_time - start_time) / LATENCY_BIN_SIZE;
      dur = dur < 256 ? dur : 255;
      s_tvalue[k] = (uint8_t)dur;

      // s_tvalue[ITER_SIZE - 1] = end_time - start_time;
      //
      // // 4 bit latency bin
      // unsigned int latency_bin = s_tvalue[ITER_SIZE - 1];
      // // unsigned int latency_bin = end_time - start_time;
      // latency_bin = (latency_bin / LATENCY_BIN_SIZE) % LATENCY_BIN_COUNT;
      // // assert(latency_bin >= 1);
      // // assert(((135 / LATENCY_BIN_SIZE) % LATENCY_BIN_COUNT) == 1);
      // const size_t tvalue_idx = k / LATENCIES_PER_TVALUE;
      // const size_t tvalue_offset = k % LATENCIES_PER_TVALUE;
      // const size_t latency_mask = (1 << LATENCY_BIN_BITS) - 1;
      // // printf("k=%u t_idx=%lu t_offset=%lu\n", k, tvalue_idx,
      // tvalue_offset);
      // // assert(latency_mask == 0xF);
      // // clear out the old bits
      // s_tvalue[tvalue_idx] &=
      //     ~(latency_mask << (tvalue_offset * LATENCY_BIN_BITS));
      // // set the new bits
      // // assert((latency_bin & ~latency_mask) == 0);
      // s_tvalue[tvalue_idx] |= (latency_bin & latency_mask)
      //                         << (tvalue_offset * LATENCY_BIN_BITS);
    } else {
      j = array[j];
    }
  }

  array[array_length] = j;
  array[array_length + 1] = array[j];

  for (size_t k = 0; k < ITER_SIZE_COMPRESSED; k++) {
    // index[k] = s_index[k];
    duration[k] = s_tvalue[k];
  }
}

__global__ void
global_latency_l1_readonly(const unsigned int *__restrict__ array,
                           int array_length, unsigned int *duration,
                           unsigned int *index, size_t warmup_iterations) {
  unsigned int start_time, end_time;
  uint32_t j = threadIdx.x;

  __shared__ uint32_t s_tvalue[ITER_SIZE];
  __shared__ uint32_t s_index[ITER_SIZE];

  for (size_t k = 0; k < ITER_SIZE; k++) {
    s_index[k] = 0;
    s_tvalue[k] = 0;
  }

  for (int it = (int)warmup_iterations * -ITER_SIZE; it < ITER_SIZE; it++) {
    if (it >= 0) {
      int k = it * blockDim.x + threadIdx.x;
      start_time = clock();
      j = __ldg(&array[j]);
      s_index[k] = j;
      end_time = clock();

      s_tvalue[k] = end_time - start_time;
    } else {
      j = __ldg(&array[j]);
    }
  }

  // array[array_length] = j;
  // array[array_length + 1] = array[j];

  for (size_t it = 0; it < ITER_SIZE; it++) {
    int k = it * blockDim.x + threadIdx.x;
    index[k] = s_index[k];
    duration[k] = s_tvalue[k];
  }
}

void parametric_measure_global(size_t N, size_t stride,
                               size_t warmup_iterations) {
  hipDeviceReset();

  // print CSV header
  fprintf(stdout, "index,latency\n");

  // allocate arrays on CPU
  unsigned int *h_a;
  h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N + 2));

  // allocate arrays on GPU
  unsigned int *d_a;
  CUDA_SAFECALL(hipMalloc((void **)&d_a, sizeof(unsigned int) * (N + 2)));

  // initialize array elements on CPU with pointers into d_a
  for (size_t i = 0; i < N; i++) {
    // original:
    h_a[i] = (i + stride) % N;
  }

  h_a[N] = 0;
  h_a[N + 1] = 0;

  // copy array elements from CPU to GPU
  CUDA_SAFECALL(
      hipMemcpy(d_a, h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice));

  size_t iter_size = USE_COMPRESSION ? ITER_SIZE_COMPRESSED : ITER_SIZE;

  unsigned int *h_index =
      (unsigned int *)malloc(sizeof(unsigned int) * iter_size);
  unsigned int *h_timeinfo =
      (unsigned int *)malloc(sizeof(unsigned int) * iter_size);

  unsigned int *duration;
  CUDA_SAFECALL(
      hipMalloc((void **)&duration, sizeof(unsigned int) * iter_size));

  unsigned int *d_index;
  CUDA_SAFECALL(
      hipMalloc((void **)&d_index, sizeof(unsigned int) * iter_size));

  hipDeviceSynchronize();
  // launch kernel
  dim3 block_dim = dim3(1);
  dim3 grid_dim = dim3(1, 1, 1);

  if (USE_COMPRESSION) {
    CUDA_SAFECALL((global_latency_compressed<<<grid_dim, block_dim>>>(
        d_a, N, duration, d_index, warmup_iterations)));
  } else {
    CUDA_SAFECALL((global_latency_l1_data<<<grid_dim, block_dim>>>(
        d_a, N, duration, d_index, warmup_iterations)));
  }

  hipDeviceSynchronize();

  CUDA_SAFECALL(hipGetLastError());

  // copy results from GPU to CPU
  hipDeviceSynchronize();

  CUDA_SAFECALL(hipMemcpy((void *)h_timeinfo, (void *)duration,
                           sizeof(unsigned int) * iter_size,
                           hipMemcpyDeviceToHost));
  CUDA_SAFECALL(hipMemcpy((void *)h_index, (void *)d_index,
                           sizeof(unsigned int) * iter_size,
                           hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  if (USE_COMPRESSION) {
    unsigned int j = 0;
    // fprintf(stdout, "%4d to %4d\n", (int)warmup_iterations * -(int)iter_size,
    //         (int)iter_size);
    for (int k = (int)warmup_iterations * -(int)iter_size; k < (int)iter_size;
         k++) {
      if (k >= 0) {
        j = h_a[j];
        unsigned int index = j;
        unsigned int binned_latency = h_timeinfo[k] * LATENCY_BIN_SIZE;
        fprintf(stdout, "%4d,%4d\n", index, binned_latency);
      } else {
        j = h_a[j];
      }
    }
  } else {
    for (size_t k = 0; k < iter_size; k++) {
      unsigned int index = h_index[k];
      unsigned int latency = h_timeinfo[k];
      fprintf(stdout, "%4d,%4d\n", index, latency);
    }
  }
  // unsigned int j = 0;
  // for (size_t k = 0; k < iter_size ; k++) {
  //   // print as CSV to stdout
  //   unsigned int index = h_index[k];
  //   unsigned int latency;
  //   if (USE_COMPRESSION) {
  //     latency = h_timeinfo[k];
  //     j = d_a[j];
  //     index = j;
  //
  //     // size_t tvalue_idx = i / LATENCIES_PER_TVALUE;
  //     // size_t tvalue_offset = i % LATENCIES_PER_TVALUE;
  //     // size_t latency_mask = (1 << LATENCY_BIN_BITS) - 1;
  //     // assert(latency_mask == 0xF);
  //     // latency = h_timeinfo[tvalue_idx] & (latency_mask << tvalue_offset);
  //     // latency = latency >> tvalue_offset;
  //     // latency = latency * LATENCY_BIN_SIZE;
  //   } else {
  //     latency = h_timeinfo[k];
  //   }
  //
  //   fprintf(stdout, "%4d,%4d\n", index, latency);
  // }

  // free memory on GPU
  hipFree(d_a);
  hipFree(d_index);
  hipFree(duration);

  // free memory on CPU
  free(h_a);
  free(h_index);
  free(h_timeinfo);

  hipDeviceReset();
}

int main(int argc, char *argv[]) {
  hipSetDevice(0);
  size_t size_bytes, stride_bytes, warmup_iterations;

  // parse arguments
  if (argc > 2) {
    size_bytes = atoi(argv[1]);
    stride_bytes = atoi(argv[2]);
    warmup_iterations = atoi(argv[3]);
  } else {
    fprintf(stderr,
            "usage: p_chase_l1 <SIZE_BYTES> <STRIDE_BYTES> <WARMUP> \n");
    return EXIT_FAILURE;
  }

  size_t iter_size = USE_COMPRESSION ? ITER_SIZE_COMPRESSED : ITER_SIZE;

  // the number of resulting patterns P (full iterations through size) is
  // P = iter_size / stride
  float one_round = (float)size_bytes / (float)stride_bytes;
  float num_rounds = (float)iter_size / one_round;

  size_t size = size_bytes / sizeof(uint32_t);
  size_t stride = stride_bytes / sizeof(uint32_t);

  fprintf(stderr,
          "\tSIZE               = %10lu bytes (%10lu uint32, %10.4f KB)\n",
          size_bytes, size, (float)size_bytes / 1024.0);
  fprintf(stderr, "\tSTRIDE             = %10lu bytes (%10lu uint32)\n ",
          stride_bytes, stride);
  fprintf(stderr, "\tROUNDS             = %3.3f\n", num_rounds);
  fprintf(stderr, "\tONE ROUND          = %3.3f (have %5lu)\n", one_round,
          iter_size);
  fprintf(stderr, "\tITERATIONS         = %lu\n", iter_size);
  fprintf(stderr, "\tWARMUP ITERATIONS  = %lu\n", warmup_iterations);

  // assert(num_rounds > 1 &&
  //        "array size is too big (rounds should be at least two)");
  // assert(iter_size > size / stride);

  // validate parameters
  if (size < stride) {
    fprintf(stderr, "ERROR: size (%lu) is smaller than stride (%lu)\n", size,
            stride);
    fflush(stderr);
    return EXIT_FAILURE;
  }
  // if (size % stride != 0) {
  //   fprintf(stderr,
  //           "ERROR: size (%lu) is not an exact multiple of stride (%lu)\n",
  //           size, stride);
  //   fflush(stderr);
  //   return EXIT_FAILURE;
  // }
  if (size < 1) {
    fprintf(stderr, "ERROR: size is < 1 (%lu)\n", size);
    fflush(stderr);
    return EXIT_FAILURE;
  }
  if (stride < 1) {
    fprintf(stderr, "ERROR: stride is < 1 (%lu)\n", stride);
    fflush(stderr);
    return EXIT_FAILURE;
  }

  // printf("\n=====%10.4f KB array, warm TLB, read iter_size element====\n",
  //          sizeof(unsigned int) * (float)N / 1024);
  //   printf("Stride = %d element, %d byte\n", stride,
  //          stride * sizeof(unsigned int));

  // The `cudaDeviceSetCacheConfig` function can be used to set preference for
  // shared memory or L1 cache globally for all CUDA kernels in your code and
  // even those used by Thrust.
  // The option cudaFuncCachePreferShared prefers shared memory, that is,
  // it sets 48 KB for shared memory and 16 KB for L1 cache.
  //
  // `cudaFuncCachePreferL1` prefers L1, that is, it sets 16 KB for
  // shared memory and 48 KB for L1 cache.
  //
  // `cudaFuncCachePreferNone` uses the preference set for the device or
  // thread.

  hipFuncCache_t want_cache_config = hipFuncCachePreferShared;
  // cudaDeviceSetCacheConfig(cudaFuncCachePreferL1);
  CUDA_SAFECALL(hipDeviceSetCacheConfig(want_cache_config));
  CUDA_SAFECALL(
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(global_latency_compressed), want_cache_config));
  hipFuncCache_t have_cache_config;
  CUDA_SAFECALL(hipDeviceGetCacheConfig(&have_cache_config));
  assert(want_cache_config == have_cache_config);

  // CUDA_SAFECALL(cudaFuncSetAttribute(
  //     global_latency_compressed, cudaFuncAttributeMaxDynamicSharedMemorySize,
  //     12 * 1024));

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  fprintf(stderr, "\tSHMEM PER BLOCK   = %lu\n", prop.sharedMemPerBlock);
  fprintf(stderr, "\tSHMEM PER SM      = %lu\n",
          prop.sharedMemPerMultiprocessor);
  fprintf(stderr, "\tL2 size           = %u\n", prop.l2CacheSize);

  parametric_measure_global(size, stride, warmup_iterations);

  hipDeviceReset();
  fflush(stdout);
  fflush(stderr);
  return EXIT_SUCCESS;
}
