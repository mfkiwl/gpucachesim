
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_SAFECALL(call)                                                {\
        call;                                                               \
        hipError_t err = hipGetLastError();                                 \
        if (hipSuccess != err) {                                           \
            fprintf(                                                        \
                stderr,                                                     \
                "Cuda error in function '%s' file '%s' in line %i : %s.\n", \
                #call, __FILE__, __LINE__, hipGetErrorString(err));        \
            fflush(stderr);                                                 \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }


// CUDA kernel. Each thread takes care of one element of c
template<typename T>
__global__ void vecAdd(T *a, T *b, T *c, int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) c[id] = a[id] + b[id];
}

template<typename T>
int vectoradd(int n) {
    // Host input vectors
    T *h_a;
    T *h_b;
    // Host output vector
    T *h_c;

    // Device input vectors
    T *d_a;
    T *d_b;
    // Device output vector
    T *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(T);

    // Allocate memory for each vector on host
    h_a = (T *)malloc(bytes);
    h_b = (T *)malloc(bytes);
    h_c = (T *)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for (i = 0; i < n; i++) {
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
        h_c[i] = 0;
    }

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n / blockSize);

    // Execute the kernel
    CUDA_SAFECALL((vecAdd<T><<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));

    // Copy array back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Sum up vector c and print result divided by n, this should equal 1 within
    // error
    T sum = 0;
    for (i = 0; i < n; i++) sum += h_c[i];
    printf("Final sum = %f; sum/n = %f (should be ~1)\n", sum, sum / n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

int main(int argc, char *argv[]) {
    // Size of vectors
    int n = 100; // used to be 100 000
    bool use_double = false;
    if (argc > 2) {
      n = atoi(argv[1]);
      if (atoi(argv[2]) == 64) use_double = true;
    } else {
      fprintf(stderr, "usage: vectoradd <n> <datatype>\n");
      return 1;
    }

    if (use_double) {
      return vectoradd<double>(n);
    } else {
      return vectoradd<float>(n);
    }
}
