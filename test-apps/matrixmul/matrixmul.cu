#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#include <hip/hip_runtime.h>

// the number of threads per block
#define BLOCK_SIZE 32

double my_timer() {
  struct timeval time;
  double _ret_val_0;
  gettimeofday((&time), 0);
  _ret_val_0 = (time.tv_sec + (time.tv_usec / 1000000.0));
  return _ret_val_0;
}

template <typename T> void mult(T *A, T *B, T *C, int size) {
  int i, j, k;
  T sum = 0.0;

  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      for (k = 0; k < size; k++) {
        sum += A[i * size + k] * B[k * size + j];
      }

      C[i * size + j] = sum;
      sum = 0.0;
    }
  }
}

template <typename T>
__global__ void mult_gpu(T *A, T *B, T *C, int wA, int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  // float Csub[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  // float Csub[8] = {0, 0, 0, 0, 0, 0, 0, 0};
  // float Csub[4] = {0, 0, 0, 0};
  // float Csub[2] = {0, 0};
  T Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix

  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ T As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ T Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];
    /*
            As[ty + 8][tx] = A[a + wA * (ty + 8) + tx];
            Bs[ty + 8][tx] = B[b + wB * (ty + 8) + tx];

            As[ty + 16][tx] = A[a + wA * (ty + 16) + tx];
            Bs[ty + 16][tx] = B[b + wB * (ty + 16) + tx];

            As[ty + 24][tx] = A[a + wA * (ty + 24) + tx];
            Bs[ty + 24][tx] = B[b + wB * (ty + 24) + tx];

            As[ty + 32][tx] = A[a + wA * (ty + 32) + tx];
            Bs[ty + 32][tx] = B[b + wB * (ty + 32) + tx];

            As[ty + 40][tx] = A[a + wA * (ty + 40) + tx];
            Bs[ty + 40][tx] = B[b + wB * (ty + 40) + tx];

            As[ty + 48][tx] = A[a + wA * (ty + 48) + tx];
            Bs[ty + 48][tx] = B[b + wB * (ty + 48) + tx];

            As[ty + 56][tx] = A[a + wA * (ty + 56) + tx];
            Bs[ty + 56][tx] = B[b + wB * (ty + 56) + tx];

            As[ty + 64][tx] = A[a + wA * (ty + 64) + tx];
            Bs[ty + 64][tx] = B[b + wB * (ty + 64) + tx];

            As[ty + 72][tx] = A[a + wA * (ty + 72) + tx];
            Bs[ty + 72][tx] = B[b + wB * (ty + 72) + tx];

            As[ty + 80][tx] = A[a + wA * (ty + 80) + tx];
            Bs[ty + 80][tx] = B[b + wB * (ty + 80) + tx];

            As[ty + 88][tx] = A[a + wA * (ty + 88) + tx];
            Bs[ty + 88][tx] = B[b + wB * (ty + 88) + tx];

            As[ty + 96][tx] = A[a + wA * (ty + 96) + tx];
            Bs[ty + 96][tx] = B[b + wB * (ty + 96) + tx];

            As[ty + 104][tx] = A[a + wA * (ty + 104) + tx];
            Bs[ty + 104][tx] = B[b + wB * (ty + 104) + tx];

            As[ty + 112][tx] = A[a + wA * (ty + 112) + tx];
            Bs[ty + 112][tx] = B[b + wB * (ty + 112) + tx];

            As[ty + 120][tx] = A[a + wA * (ty + 120) + tx];
            Bs[ty + 120][tx] = B[b + wB * (ty + 120) + tx];
    */

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
      /*
                  Csub[0] += As[ty][k] * Bs[k][tx];
                  Csub[1] += As[ty + 8][k] * Bs[k][tx];

                  Csub[2] += As[ty + 16][k] * Bs[k][tx];
                  Csub[3] += As[ty + 24][k] * Bs[k][tx];

                  Csub[4] += As[ty + 32][k] * Bs[k][tx];
                  Csub[5] += As[ty + 40][k] * Bs[k][tx];

                  Csub[6] += As[ty + 48][k] * Bs[k][tx];
                  Csub[7] += As[ty + 56][k] * Bs[k][tx];

                  Csub[8] += As[ty + 64][k] * Bs[k][tx];
                  Csub[9] += As[ty + 72][k] * Bs[k][tx];

                  Csub[10] += As[ty + 80][k] * Bs[k][tx];
                  Csub[11] += As[ty + 88][k] * Bs[k][tx];

                  Csub[12] += As[ty + 96][k] * Bs[k][tx];
                  Csub[13] += As[ty + 104][k] * Bs[k][tx];

                  Csub[14] += As[ty + 112][k] * Bs[k][tx];
                  Csub[15] += As[ty + 120][k] * Bs[k][tx];
      */
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;

  /*
      C[c + wB * ty + tx] = Csub[0];
      C[c + wB * (ty + 8) + tx] = Csub[1];

      C[c + wB * (ty + 16) + tx] = Csub[2];
      C[c + wB * (ty + 24) + tx] = Csub[3];

      C[c + wB * (ty + 32) + tx] = Csub[4];
      C[c + wB * (ty + 40) + tx] = Csub[5];

      C[c + wB * (ty + 48) + tx] = Csub[6];
      C[c + wB * (ty + 56) + tx] = Csub[7];

      C[c + wB * (ty + 64) + tx] = Csub[8];
      C[c + wB * (ty + 72) + tx] = Csub[9];

      C[c + wB * (ty + 80) + tx] = Csub[10];
      C[c + wB * (ty + 88) + tx] = Csub[11];

      C[c + wB * (ty + 96) + tx] = Csub[12];
      C[c + wB * (ty + 104) + tx] = Csub[13];

      C[c + wB * (ty + 112) + tx] = Csub[14];
      C[c + wB * (ty + 120) + tx] = Csub[15];
  */

  // __threadfence_system();
}

template <typename T> int matrixmul(int MROW) {
  int i;
  T *A, *B, *C, *D;
  T *A_dev, *B_dev, *C_dev;
  double start_timer, end_timer;

  int MSIZE = MROW * MROW;
  printf("(%d x %d) x (%d x %d)\n", MROW, MROW, MROW, MROW);
  printf("data type: %lu bytes (%lu bits)\n", sizeof(T), sizeof(T) * 8);

  A = (T *)malloc(sizeof(T) * MSIZE);
  hipMalloc(&A_dev, MSIZE * sizeof(T));
  B = (T *)malloc(sizeof(T) * MSIZE);
  hipMalloc(&B_dev, MSIZE * sizeof(T));
  C = (T *)malloc(sizeof(T) * MSIZE);
  hipMalloc(&C_dev, MSIZE * sizeof(T));
  D = (T *)malloc(sizeof(T) * MSIZE);

  srand(time(NULL));
  // Init matrix
  for (i = 0; i < MSIZE; i++) {
    // A[i] = (i%MROW)+1;
    A[i] = ((T)rand() / (RAND_MAX)) + 1;
    // B[i] = (i%MCOL)+1;
    B[i] = ((T)rand() / (RAND_MAX)) + 1;
    C[i] = 0;
    D[i] = 0;
  }

  // transfer data to device
  hipMemcpy(A_dev, A, MSIZE * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(B_dev, B, MSIZE * sizeof(T), hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  dim3 threads(BLOCK_SIZE, BLOCK_SIZE / 1);
  int grid_size = (MROW + (BLOCK_SIZE - 1)) / BLOCK_SIZE;
  dim3 grid(grid_size, grid_size);
  printf("grid: (%d,%d,%d)\n", grid.x, grid.y, grid.z);
  printf("threads: (%d,%d,%d)\n", threads.x, threads.y, threads.z);

  assert(grid.x > 0);
  assert(grid.y > 0);
  assert(grid.z > 0);

  /* printf("block:%d, thread:%d\n", (MROW / BLOCK_SIZE) * (MROW / BLOCK_SIZE),
   */
  /*        BLOCK_SIZE * BLOCK_SIZE); */
  start_timer = my_timer();
  mult_gpu<T><<<grid, threads, 0>>>(A_dev, B_dev, C_dev, MROW, MROW);
  hipDeviceSynchronize();
  end_timer = my_timer();
  printf("The GPU Elapsed Time:%lf Sec.\n", end_timer - start_timer);

  // transfer data back to host
  hipMemcpy(C, C_dev, MSIZE * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  start_timer = my_timer();
  mult<T>(A, B, D, MROW);
  end_timer = my_timer();
  printf("The CPU Elapsed Time:%lf Sec.\n", end_timer - start_timer);

  // Verification
  printf("Verifying\n");
  bool correct = true;
  for (i = 0; i < MSIZE; i++) {
    if (abs(C[i] - D[i]) > 1e-2) {
      printf("Error:%f, %f\n", C[i], D[i]);
      correct = false;
      break;
    }
  }
  if (correct) {
    printf("PASS\n");
  }

  free(A);
  hipFree(A_dev);
  free(B);
  hipFree(B_dev);
  free(C);
  hipFree(C_dev);
  free(D);
  return 0;
}

int main(int argc, char *argv[]) {
  if (argc != 3) {
    fprintf(stderr, "usage: matrixmul <mrow> <datatype>\n");
    return 1;
  }
  int MROW = atoi(argv[1]);
  if (MROW < 32) {
    fprintf(stderr,
            "ERROR: matrices with less than 32 rows are not supported\n");
    return 1;
  }
  bool use_double = (atoi(argv[2]) == 64);
  if (use_double) {
    return matrixmul<double>(MROW);
  } else {
    return matrixmul<float>(MROW);
  }
}
